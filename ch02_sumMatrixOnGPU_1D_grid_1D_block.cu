#include<hip/hip_runtime.h>

#include<stdio.h>
#include<cmath>
#include<time.h>
#include<string.h>

#define CHECK(call){	\
const hipError_t error = call;		\
if (error!=hipSuccess) {	\
	printf("Error:%s:%d, ", __FILE__, __LINE__);	\
	printf("code:%d, reason: %s\n", error,	\
			hipGetErrorString(error));	\
	exit(1);	\
	}	\
}	\

void printMatrix(int *C, const int nx, const int ny) {
	int *ic = C;
	printf("\nMatrix: (%d.%d)\n", nx, ny);
	for (int iy = 0; iy<ny; iy++) {
		for (int ix = 0; ix<nx; ix++) {
			printf("%3d", ic[ix]);
		}
		ic += nx;
		printf("\n");
	}
	printf("\n");
	return;
}

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny) {
	/*
	int ix = threadIdx.x+blockIdx.x*blockDim.x;
	int iy = threadIdx.y+blockIdx.y*blockDim.y;
	unsigned int idx = iy*nx+ix;
	*/

	unsigned int ix = threadIdx.x+blockIdx.x*blockDim.x;


	if (ix<nx) {
		for (int iy = 0; iy<ny; iy++) {
			int idx = iy*nx+ix;
			MatC[idx] = MatA[idx]+MatB[idx];
		}
	}
}

void initialData(float *ip, int size) {
	//乱数シード生成
	time_t t;
	srand((unsigned)time(&t));

	for (int i = 0; i<size; i++) {
		ip[i] = (float)(rand()&0xFF)/10.0f;
	}
	return;
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny) {
	float *ia = A;
	float *ib = B;
	float *ic = C;

	for (int iy = 0; iy<ny; iy++) {
		for (int ix = 0; ix<nx; ix++) {
			ic[ix] = ia[ix]+ib[ix];
		}
		ia += nx;
		ib += nx;
		ic += nx;
	}
	return;
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
	double epsilon = 1.0E-8;
	bool match = 1;

	for (int i = 0; i<N; i++) {
		if (abs(hostRef[i]-gpuRef[i])>epsilon) {
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpur %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
			break;
		}
	}
}

int main(int argc, char **argv) {
	printf("%s Starting...\n", argv[0]);

	//デバイスのセットアップ
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	//行列のデータサイズを設定
	int nx = 1<<14;
	int ny = 1<<14;

	int nxy = nx*ny;
	int nBytes = nxy*sizeof(float);
	printf("Matrix size: nx%d ny%d\n", nx, ny);

	//ホストメモリ確保
	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	clock_t iStart;



	//ホスト側でデータを初期化
	iStart = clock();
	initialData(h_A, nxy);
	initialData(h_B, nxy);
	double iElaps = clock()-iStart;



	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);



	//結果をチェックするためにホスト側で行列を加算
	iStart = clock();
	sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
	iElaps = clock()-iStart;


	//デバイスのグローバルメモリを確保
	float *d_MatA, *d_MatB, *d_MatC;
	CHECK(hipMalloc((void **)&d_MatA, nBytes));
	CHECK(hipMalloc((void **)&d_MatB, nBytes));
	CHECK(hipMalloc((void **)&d_MatC, nBytes));

	//ホストからデバイスデータを転送
	CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));


	//ホスト側でカーネルを呼び出す
	//dim3 block(dimx, dimy);
	dim3 block(32, 1);
	//dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);
	dim3 grid((nx+block.x-1)/block.x, 1);

	iStart = clock();
	//sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
	sumMatrixOnGPU1D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
	CHECK(hipDeviceSynchronize());
	iElaps = clock()-iStart;
	printf("sumMatrixOnGPU1D<<<(%d,%d),(%d,%d)>>>elapsed %f sec\n", grid.x, grid.y, block.x, block.y, iElaps/CLOCKS_PER_SEC);

	//カーネルエラーチェック
	CHECK(hipGetLastError());

	//カーネルの結果をホスト側にコピー
	CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

	//デバイスの結果をチェック
	checkResult(hostRef, gpuRef, nxy);

	//デバイスのグローバルメモリを解放
	CHECK(hipFree(d_MatA));
	CHECK(hipFree(d_MatB));
	CHECK(hipFree(d_MatC));

	//ホストのメモリを解放
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	//デバイスリセット
	CHECK(hipDeviceReset());

	return 0;
}
