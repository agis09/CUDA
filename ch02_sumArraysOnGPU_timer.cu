#include<hip/hip_runtime.h>

#include<stdio.h>
#include<cmath>
#include<time.h>
#include<string.h>
#include<time.h>

#define CHECK(call){	\
const hipError_t error = call;		\
if (error!=hipSuccess) {	\
	printf("Error:%s:%d, ", __FILE__, __LINE__);	\
	printf("code:%d, reason: %s\n", error,	\
			hipGetErrorString(error));	\
	exit(1);	\
	}	\
}	\

void checkResult(float *hostRef, float *gpuRef, const int N) {
double epsilon = 1.0E-8;
bool match = 1;

for (int i = 0; i<N; i++) {
	if (abs(hostRef[i]-gpuRef[i])>epsilon) {
		match = 0;
		printf("Arrays do not match!\n");
		printf("host %5.2f gpur %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
		break;
	}
}
if (match)printf("Arrays match.\n\n");
return;
}

void initialData(float *ip, int size) {
//�����V�[�h����
time_t t;
srand((unsigned)time(&t));

for (int i = 0; i<size; i++) {
	ip[i] = (float)(rand()&0xFF)/10.0f;
}
return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
	for (int idx = 0; idx<N; idx++) {
		C[idx] = A[idx]+B[idx];
	}
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	C[i] = A[i]+B[i];
}


int main(int argc, char **argv) {
	printf("%s Starting...\n", argv[0]);

	//�f�o�C�X�̃Z�b�g�A�b�v
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	//�x�N�g���̃f�[�^�T�C�Y��ݒ�
	int nElem = 1<<24;
	printf("Vector size %d\n", nElem);

	//�z�X�g�������m��
	size_t nBytes = nElem*sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);
	clock_t iStart;
	double iElaps;
	
	//�z�X�g���Ńf�[�^��������
	iStart = clock();
	initialData(h_A, nElem);
	initialData(h_B, nElem);
	iElaps = clock()-iStart;
	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	//���ʂ��`�F�b�N���邽�߂Ƀz�X�g���Ńx�N�g�����Z
	iStart = clock();
	sumArraysOnHost(h_A, h_B, hostRef, nElem);
	iElaps = clock()-iStart;


	//�f�o�C�X�̃O���[�o���������m��
	float *d_A, *d_B, *d_C;
	hipMalloc((float**)&d_A, nBytes);
	hipMalloc((float**)&d_B, nBytes);
	hipMalloc((float**)&d_C, nBytes);

	//�z�X�g����f�o�C�X�փf�[�^�]��
	CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

	//�z�X�g���ŃJ�[�l�����Ăяo��
	int iLen = 512;	//1024->512	
	dim3 block(iLen);
	dim3 grid((nElem+block.x-1)/block.x);

	iStart = clock();
	sumArraysOnGPU<<< grid, block>>>(d_A, d_B, d_C, nElem);
	CHECK(hipDeviceSynchronize());
	iElaps = clock()-iStart;
	printf("sumArraysOnGPU <<<%d, %d>>> Time elapsed %fsec\n", grid.x, block.x,iElaps/CLOCKS_PER_SEC);

	//�J�[�l���̃G���[�`�F�b�N
	CHECK(hipGetLastError());

	//�J�[�l���̌��ʂ��z�X�g���ɃR�s�[
	CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));


	//�f�o�C�X�̌��ʂ��`�F�b�N
	checkResult(hostRef, gpuRef, nElem);

	//�f�o�C�X�̃O���[�o�����������
	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));

	//�z�X�g�̃��������
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	//cudaDeviceReset();
	return 0;

}
