
#include <hip/hip_runtime.h>
/*
#include<cuda_runtime.h>
#include<device_launch_parameters.h>
#include<stdio.h>
#include<cmath>
//#include<time.h>
//#include<string.h>

#define CHECK(call){	\
const cudaError_t error = call;		\
if (error!=cudaSuccess) {	\
	printf("Error:%s:%d, ", __FILE__, __LINE__);	\
	printf("code:%d, reason: %s\n", error,	\
			cudaGetErrorString(error));	\
	exit(1);	\
	}	\
}	\

void printMatrix(int *C, const int nx, const int ny) {
	int *ic = C;
	printf("\nMatrix: (%d.%d)\n", nx, ny);
	for (int iy = 0; iy<ny; iy++) {
		for (int ix = 0; ix<nx; ix++) {
			printf("%3d", ic[ix]);
		}
		ic += nx;
		printf("\n");
	}
	printf("\n");
	return;
}

__global__ void printThreadIndex(int *A, const int nx, const int ny) {
	int ix = threadIdx.x+blockIdx.x*blockDim.x;
	int iy = threadIdx.y+blockIdx.y*blockDim.y;
	unsigned int idx = iy*nx+ix;

	printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d %d) global index %2d ival %2d\n",
		threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv) {
	printf("%s Starting... \n", argv[0]);

	//デバイス情報取得
	int dev = 0;
	cudaDeviceProp deviceProp;
	CHECK(cudaGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(cudaSetDevice(dev));

	//行列の次元を設定
	int nx = 8;
	int ny = 6;
	int nxy = nx*ny;
	int nBytes = nxy*sizeof(float);
	//ホストメモリ確保
	int *h_A;
	h_A = (int *)malloc(nBytes);

	//ホスト行列を整数で初期化
	for (int i = 0; i<nxy; i++) {
		h_A[i] = i;
	}
	printMatrix(h_A, nx, ny);

	//デバイスメモリを確保
	int *d_MatA;
	CHECK(cudaMalloc((void **)&d_MatA,nBytes));

	//ホストからデバイスへデータを転送
	CHECK(cudaMemcpy(d_MatA, h_A, nBytes, cudaMemcpyHostToDevice));

	//実行設定をセットアップ
	dim3 block(4, 2);
	dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

	//カーネルを呼び出す
	printThreadIndex<<<grid, block>>>(d_MatA, nx, ny);
	CHECK(cudaDeviceSynchronize());

	//ホストとデバイスのメモリを解放
	CHECK(cudaFree(d_MatA));
	free(h_A);

	//デバイスをリセット
	CHECK(cudaDeviceReset());

	return 0;
}
*/