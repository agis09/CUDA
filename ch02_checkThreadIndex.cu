
#include <hip/hip_runtime.h>
/*
#include<cuda_runtime.h>
#include<device_launch_parameters.h>
#include<stdio.h>
#include<cmath>
//#include<time.h>
//#include<string.h>

#define CHECK(call){	\
const cudaError_t error = call;		\
if (error!=cudaSuccess) {	\
	printf("Error:%s:%d, ", __FILE__, __LINE__);	\
	printf("code:%d, reason: %s\n", error,	\
			cudaGetErrorString(error));	\
	exit(1);	\
	}	\
}	\

void printMatrix(int *C, const int nx, const int ny) {
	int *ic = C;
	printf("\nMatrix: (%d.%d)\n", nx, ny);
	for (int iy = 0; iy<ny; iy++) {
		for (int ix = 0; ix<nx; ix++) {
			printf("%3d", ic[ix]);
		}
		ic += nx;
		printf("\n");
	}
	printf("\n");
	return;
}

__global__ void printThreadIndex(int *A, const int nx, const int ny) {
	int ix = threadIdx.x+blockIdx.x*blockDim.x;
	int iy = threadIdx.y+blockIdx.y*blockDim.y;
	unsigned int idx = iy*nx+ix;

	printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d %d) global index %2d ival %2d\n",
		threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv) {
	printf("%s Starting... \n", argv[0]);

	//�f�o�C�X���擾
	int dev = 0;
	cudaDeviceProp deviceProp;
	CHECK(cudaGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(cudaSetDevice(dev));

	//�s��̎�����ݒ�
	int nx = 8;
	int ny = 6;
	int nxy = nx*ny;
	int nBytes = nxy*sizeof(float);
	//�z�X�g�������m��
	int *h_A;
	h_A = (int *)malloc(nBytes);

	//�z�X�g�s��𐮐��ŏ�����
	for (int i = 0; i<nxy; i++) {
		h_A[i] = i;
	}
	printMatrix(h_A, nx, ny);

	//�f�o�C�X���������m��
	int *d_MatA;
	CHECK(cudaMalloc((void **)&d_MatA,nBytes));

	//�z�X�g����f�o�C�X�փf�[�^��]��
	CHECK(cudaMemcpy(d_MatA, h_A, nBytes, cudaMemcpyHostToDevice));

	//���s�ݒ���Z�b�g�A�b�v
	dim3 block(4, 2);
	dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

	//�J�[�l�����Ăяo��
	printThreadIndex<<<grid, block>>>(d_MatA, nx, ny);
	CHECK(cudaDeviceSynchronize());

	//�z�X�g�ƃf�o�C�X�̃����������
	CHECK(cudaFree(d_MatA));
	free(h_A);

	//�f�o�C�X�����Z�b�g
	CHECK(cudaDeviceReset());

	return 0;
}
*/